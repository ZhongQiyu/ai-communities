#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;

    __shared__ float r[_N_], k[_N_];

    float state[_N_] = {0};

    for (int _t = b*T*C + h*_N_ + i; _t < (b+1)*T*C + h*_N_ + i; _t += C)
    {
        __syncthreads();
        r[i] = float(_r[_t]);
        k[i] = float(_k[_t]);
        __syncthreads();

        const float v = float(_v[_t]);
        float y = 0;

        for (int j = 0; j < _N_; j++)
        {
            float x = k[j] * v;

            float s = state[j];
            state[j] = s * _w[j] + x;

            y += r[j] * (float(_u[j]) * x + s);
        }
        _y[_t] = F(y);
    }
}

template <typename F>
__global__ void kernel_backward(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const float *__restrict__ __w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gr, F *__restrict__ const _gk, F *__restrict__ const _gv, F *__restrict__ const _gw, F *__restrict__ const _gu)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;
    __w += h*_N_;
    const float w = _w[i];
    const float u = float(_u[i]);
    const float ww = __w[i];

    __shared__ float v[_N_], r[_N_], k[_N_], gy[_N_], gy2[_N_], w_[_N_], u_[_N_];    
    float state[_N_] = {0}, saaaa[_N_] = {0}, sbbbb[_N_] = {0};

    float gw = 0, gu = 0;
    const int t000 = b*T*C + h*_N_ + i;
    const int t111 = (b+1)*T*C + h*_N_ + i;
    const int t222 = t111 - 2*C;

    for (int _t = t000; _t < t111; _t += C)
    {
        __syncthreads();
        v[i] = float(_v[_t]);
        gy[i] = float(_gy[_t]);
        if (_t < t222)
            gy2[i] = float(_gy[_t + 2*C]);
        __syncthreads();

        const float k = float(_k[_t]);
        const float r = float(_r[_t]);
        const float r2 = (_t < t222) ? float(_r[_t + 2*C]) : 0;
        
        float gr = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float x = v[j] * k;
            float s = state[j];
            state[j] = s * w + x;

            gr += gy[j] * (u * x + s);
            gu += r * x * gy[j];
        }
        _gr[_t] = F(gr);
        
        if (_t < t222)
        {
            #pragma unroll
            for (int j = 0; j < _N_; j++)
            {
                float x = v[j] * k;
                saaaa[j] = w * (saaaa[j] + sbbbb[j] + x);
                sbbbb[j] = w * (sbbbb[j] + x);
                
                gw += r2 * ww * saaaa[j] * gy2[j];
            }
        }
    }
    _gu[b*C + h*_N_ + i] = F(gu);
    _gw[b*C + h*_N_ + i] = F(gw);

    #pragma unroll
    for (int j = 0; j < _N_; ++j) {
        saaaa[j] = 0;
        sbbbb[j] = 0;
    }

    __syncthreads();
    w_[i] = float(_w[i]);
    u_[i] = float(_u[i]);
    __syncthreads();
    
    for (int _t = t111 - C; _t >= t000; _t -= C)
    {
        __syncthreads();
        v[i] = float(_v[_t]);
        gy[i] = float(_gy[_t]);
        k[i] = float(_k[_t]);
        r[i] = float(_r[_t]);
        __syncthreads();

        float gk = 0, gv = 0, x, s;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            x = gy[j] * r[i];
            s = saaaa[j];
            saaaa[j] = s * w + x;
            gk += v[j] * (u * x + s);

            x = gy[i] * r[j];
            s = sbbbb[j];
            sbbbb[j] = s * w_[j] + x;
            gv += k[j] * (u_[j] * x + s);
        }
        _gk[_t] = F(gk);
        _gv[_t] = F(gv);
    }
}

void cuda_forward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, float *w, bf16 *u, bf16 *y)
{
    assert(H*_N_ == C);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, y);
}

void cuda_backward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, float *w, float *ww, bf16 *u, bf16 *gy, bf16 *gr, bf16 *gk, bf16 *gv, bf16 *gw, bf16 *gu)
{
    assert(H*_N_ == C);
    kernel_backward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, ww, u, gy, gr, gk, gv, gw, gu);
}
